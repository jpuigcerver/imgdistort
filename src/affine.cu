#include "hip/hip_runtime.h"
#include "affine.h"

#include <glog/logging.h>

#include "utils.h"

#define BLOCK_SIZE 16

template <typename T>
__global__
void kernel_affine_NCHW(
    T* dst, const T* src, const int N, const int C, const int H, const int W,
    const T* M, const int Mn) {
  const int Bx = BLOCK_SIZE * (blockIdx.x % DIV_UP(W, BLOCK_SIZE));
  const int By = BLOCK_SIZE * blockIdx.y;
  const int n = blockIdx.z;
  const int c = blockIdx.x / DIV_UP(W, BLOCK_SIZE);
  const int x = Bx + threadIdx.x;
  const int y = By + threadIdx.y;

  // Copy affine transformation matrix into shared memory and invert it.
  // The affine matrix needs to be inverted because the kernel actually computes
  // the inverse operation.
  // All threads in the block work on the same image, and thus all use the same
  // affine matrix.
  __shared__ T _M[6];
  const int offset_M = (n % Mn) * 6;
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    invert_affine_matrix(M + offset_M, _M);
  }
  __syncthreads();

  // Compute output pixel value
  if (x >= W || y >= H) return;
  const T rx = _M[0] * x + _M[1] * y + _M[2];
  const T ry = _M[3] * x + _M[4] * y + _M[5];
  const int offset_S = n * C * H * W + c * H * W;
  dst[offset_S + y * W + x] = blinterp(src + offset_S, rx, ry, W, H);
}

template <typename T>
void call_kernel_affine_NCHW(T* dst, const T* src,
                             const int N, const int C, const int H, const int W,
                             const T* M, const int Mn, hipStream_t stream) {
  CHECK_NOTNULL(dst);
  CHECK_NOTNULL(src);
  CHECK_GT(N, 0);
  CHECK_GT(C, 0);
  CHECK_GT(H, 0);
  CHECK_GT(W, 0);
  CHECK_NOTNULL(M);
  CHECK_GT(Mn, 0);
  const dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
  const dim3 grid_size(C * DIV_UP(W, BLOCK_SIZE), DIV_UP(H, BLOCK_SIZE), N);
  kernel_affine_NCHW<<<grid_size, block_size, 0, stream>>>(
      dst, src, N, C, H, W, M, Mn);
  CHECK_LAST_CUDA_CALL;
  if (stream == 0) {
    CHECK_CUDA_CALL(hipDeviceSynchronize());
  }
}

void affine_NCHW_f32(float* dst, const float* src,
                     const int N, const int C, const int H, const int W,
                     const float* M, const int Mn, hipStream_t stream) {
  call_kernel_affine_NCHW<float>(dst, src, N, C, H, W, M, Mn, stream);
}

void affine_NCHW_f64(double* dst, const double* src,
                     const int N, const int C, const int H, const int W,
                     const double* M, const int Mn, hipStream_t stream) {
  call_kernel_affine_NCHW<double>(dst, src, N, C, H, W, M, Mn, stream);
}
